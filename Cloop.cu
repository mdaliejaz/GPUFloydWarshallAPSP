#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>


int rowSize;

__global__ void printGpu(float *d_a, int size)
{
	   int i,j;
        for(i=0;i<size;i++)
        {
                for(j=0;j<size;j++)
                        printf("%0.1f\t", d_a[i*size+j]);
               	printf("\n");
        }
}

__global__ void Cloop_FW(float *d_a,int j,int k, int rowSize)
{
	__shared__ int intermed;
	int col =  blockIdx.x * blockDim.x + threadIdx.x;
	if(col >= rowSize)
		return;

	if (threadIdx.x == 0) {
		intermed = d_a[k*rowSize+j];
		//d_a[threadIdx.x*rowSize + j ]  = fmin( d_a[threadIdx.x*rowSize + j ], d_a[threadIdx.x*rowSize + k] + intermed); 	
	}
	__syncthreads();

        d_a[col*rowSize + j ]  = fmin( d_a[col*rowSize + j ], d_a[col*rowSize + k] + intermed);


}

void print_matrix(float *d,int size)
{
	int i,j;
	for(i=0;i<size;i++)	
	{
		for(j=0;j<size;j++)
			printf("%0.1f\t", d[i*size+j]);
		puts("");
	}
}

int main(int argc, char** argv) 
{

	float *d_a;
	float *a;
	
	size_t pitch;
	rowSize = 8192;
	int colSize = rowSize;
	int i,j,k;
	hipError_t err = hipSuccess;  
	size_t totalSize = rowSize*colSize*sizeof(float);	

	a = (float *) malloc(totalSize);
	if (!a)
	{
		printf("Unable to allocate memory for host array\n");
		return 1;
	}	
	
	err = hipMallocPitch(&d_a, &pitch, rowSize * sizeof(float), colSize);
	if(!d_a)
		{
			printf("memory failed for hipMalloc");
			return 1;
		}  
  	
	if(err !=0){
        	printf("%s-%d",hipGetErrorString(err),3);
        	//getchar();  
    	}   


	for(i = 0; i < rowSize;i++)
		for (j=0;j<colSize;j++)
		{
			if (i == j){
				a[i*rowSize+j] = 0;
			}
			else {

				a[i*rowSize+j] = (i+j)%5? (i+j) : (i+j)%7;
			}
		}
 

	//puts("input matrix :");

	//print_matrix(a,rowSize);
	
	err = hipMemcpy(d_a, a, totalSize, hipMemcpyHostToDevice);

	if(err !=0){
        	printf("after h2d %s-%d",hipGetErrorString(err),3);
        getchar();  
    	}   
	
	int threadsPerBlock;
 	int noOfBlocks;
 
 	if (rowSize < 1024)
        { 
		threadsPerBlock	= rowSize;
	}
	else
 	{
 		threadsPerBlock = 1024;
 	}
 
        noOfBlocks = rowSize / threadsPerBlock;

	for(k=0;k<rowSize;k++)
	{
	   for(j = 0; j < colSize; j++)
	   {
                	Cloop_FW<<<noOfBlocks,threadsPerBlock>>>(d_a,j,k,rowSize);
			hipDeviceSynchronize();		
	   }
	}

	printf("error = %s\n", hipGetErrorString(hipGetLastError()));

       	err = hipMemcpy(a, d_a, totalSize, hipMemcpyDeviceToHost);

	if(err !=0){
        	printf("final %s-%d",hipGetErrorString(err),3);
        //	getchar();  
    	}   

	puts("output matrix :");	
	print_matrix(a,rowSize);

	free(a);
	hipFree(d_a);
	return 0;
}

