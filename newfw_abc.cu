#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<limits.h>
#include<algorithm>
#include<sys/time.h>

using namespace std;

#define INF           INT_MAX-1

int tilesize[2] = {2, INT_MAX};
int rowSize;

void print_matrix(float *d)
{
	int i,j;
	for(i=0;i<32;i++)	
	{
		for(j=0;j<32;j++)
			printf("%0.1f\t", d[i*rowSize+j]);
		puts("");
	}
}

__global__ void Dloop_FW(float *d_a,int k, int rowSize)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col >= rowSize)
		return;

	__shared__ int intermed;
        if (threadIdx.x == 0) {
               	intermed = d_a[rowSize*blockIdx.y + k];
       	}

       __syncthreads();

        d_a[blockIdx.y*rowSize + col]  = fmin(d_a[blockIdx.y*rowSize + col], intermed + d_a[k*rowSize+col]);

}

void FW_D_loop(float* d_a,int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int size)
{        
	int threadsPerBlock;

	if (size <= 1024)
		threadsPerBlock = size;
	else
		threadsPerBlock = 1024;

	dim3 blocksPerGrid( (rowSize + threadsPerBlock -1)/threadsPerBlock ,rowSize);

	for(int k = vRowStart; k < (vRowStart + size); k++)
	{
		        Dloop_FW<<<blocksPerGrid,threadsPerBlock>>>(d_a,k,rowSize);
			hipDeviceSynchronize();

	}
}

void DFW(float *d_a,int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int currSize, int d)
{
	int r = tilesize[d];
	if (r >= currSize)
		FW_D_loop(d_a, xRowStart, xColStart, uRowStart, uColStart, vRowStart, vColStart, currSize);
	else
	{
		int newsize = currSize/r;
		for(int k=1; k<=r; k++) {	
			for(int i=1; i<=r; i++) {
				for(int j=1; j<=r; j++) {
					DFW(d_a,(i-1)*newsize, (j-1)*newsize, (i-1)*newsize, (k-1)*newsize, (k-1)*newsize, (j-1)*newsize, newsize, d+1);
				}
			}	
		}
	}
}


__global__ void Cloop_FW(float *d_a,int vRowStart,int xColStart,int size, int rowSize)
{

	__shared__ int intermed;
	int col =  blockIdx.x * blockDim.x + threadIdx.x;
	if(col >= rowSize)
		return;
	

 	for(int k = vRowStart; k < (vRowStart + size); k++)
        {

                for(int j = xColStart; j < (xColStart + size); j++)
              {  
                  
			if (threadIdx.x == 0) 
				intermed = d_a[k*rowSize+j];
		
		__syncthreads();

        	d_a[col*rowSize + j ]  = fmin( d_a[col*rowSize + j ], d_a[col*rowSize + k] + intermed);
	   }
	}

}

void FW_C_loop(float *d_a, int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int size)
{        

	int threadsPerBlock;

        if (size <= 1024)
                threadsPerBlock = size;
        else
                threadsPerBlock = 1024;

        dim3 blocksPerGrid( (rowSize + threadsPerBlock -1)/threadsPerBlock ,rowSize);
	
	Cloop_FW<<<blocksPerGrid, threadsPerBlock>>>(d_a,vRowStart,xColStart,size,rowSize);
}



void CFW(float *d_a,int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int currSize, int d)
{
	int r = tilesize[d];
	if (r >= currSize)
		FW_C_loop(d_a,xRowStart, xColStart, uRowStart, uColStart, vRowStart, vColStart, currSize);
	else
	{
		int newsize = currSize/r;
		for(int k=1; k<=r; k++) {	
			for(int i=1; i<=r; i++) {
				CFW(d_a,(i-1)*newsize, (k-1)*newsize, (i-1)*newsize, (k-1)*newsize, (k-1)*newsize, (k-1)*newsize, newsize, d+1);
			}

			for(int i=1; i<=r; i++) {
				for(int j=1; j<=r; j++) {
					if(j != k)
						DFW(d_a,(i-1)*newsize, (j-1)*newsize, (i-1)*newsize, (k-1)*newsize, (k-1)*newsize, (j-1)*newsize, newsize, d+1);
				}
			}	
		}
	}
}


__global__ void Bloop_FW(float *d_a,int i,int k, int colSize)
{


	__shared__ int intermed;
//	if (threadIdx.x == k) {
		intermed = d_a[i*colSize+k];
//		d_a[i*colSize + threadIdx.x ]  = fmin(intermed + d_a[k*colSize + threadIdx.x] ,  d_a[i*colSize+threadIdx.x ]); 	
//	}

//	 __syncthreads();

	d_a[i*colSize +	threadIdx.x ]  = fmin(intermed + d_a[k*colSize + threadIdx.x], d_a[i*colSize+threadIdx.x ]);


}

void FW_B_loop(float *d_a,int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int size)
{        

	int threadsPerBlock;
	if (size < 1024)
        { 
                threadsPerBlock	= size;
       	}
	else
	{
		threadsPerBlock = 1024;
	}

	int noOfBlocks = rowSize / threadsPerBlock;

	for(int k = vRowStart; k < (vRowStart + size); k++)
	{
		for(int i = xRowStart; i < (xRowStart + size); i++)
		{
				Bloop_FW<<<noOfBlocks,threadsPerBlock>>>(d_a,i,k,rowSize);
				hipDeviceSynchronize();

		}
	}
}

void BFW(float* d_a, int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int currSize, int d)
{
	int r = tilesize[d];
	if (r >= currSize)
		FW_B_loop(d_a,xRowStart, xColStart, uRowStart, uColStart, vRowStart, vColStart, currSize);
	else
	{
		int newsize = currSize/r;
		for(int k=1; k<=r; k++) {
			for(int j=1; j<=r; j++) {
				BFW(d_a,(k-1)*newsize, (j-1)*newsize, (k-1)*newsize, (k-1)*newsize, (k-1)*newsize, (j-1)*newsize, newsize, d+1);
			}

			for(int i=1; i<=r; i++) {
				for(int j=1; j<=r; j++) {
					if(i != k)
						DFW(d_a,(i-1)*newsize, (j-1)*newsize, (i-1)*newsize, (k-1)*newsize, (k-1)*newsize, (j-1)*newsize, newsize, d+1);
				}
			}	
		}
	}
}


__global__ void Aloop_FW(float *d_a,int rowSize)
{

	int col =  blockIdx.x * blockDim.x + threadIdx.x;
	if(col >= rowSize)
		return;

	for(int k=0;k<rowSize;k++)
        {
           for(int i = 0; i < rowSize;i++)
           {
                for (int j=0;j< rowSize;j++)
                {

		d_a[i*rowSize+j] = fmin( d_a[i*rowSize+k] + d_a[k*rowSize+j] ,d_a[i*rowSize+j]);
		}
	    }
	}

}

void FW_A_loop(float* d_a, int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int size)
{

		Aloop_FW<<<1,1>>>(d_a,rowSize);
//		hipDeviceSynchronize();
}



void AFW(float* d_a, int xRowStart, int xColStart, int uRowStart, int uColStart, int vRowStart, int vColStart, int currSize, int d)
{
	int r = tilesize[d];
	if (r >= currSize)
		FW_A_loop(d_a,xRowStart, xColStart, uRowStart, uColStart, vRowStart, vColStart, currSize);
	else
	{
		int newsize = currSize/r;
		for(int k=1; k<=r; k++) {
			AFW(d_a,(k-1)*newsize, (k-1)*newsize, (k-1)*newsize, (k-1)*newsize, (k-1)*newsize, (k-1)*newsize, newsize, d+1);
			
			for(int j=1; j<=r; j++) {
				if(j != k)
					BFW(d_a,(k-1)*newsize, (j-1)*newsize, (k-1)*newsize, (k-1)*newsize, (k-1)*newsize, (j-1)*newsize, newsize, d+1);
			}
			
			for(int i=1; i<=r; i++) {
				if(i != k)
					CFW(d_a,(i-1)*newsize, (k-1)*newsize, (i-1)*newsize, (k-1)*newsize, (k-1)*newsize, (k-1)*newsize, newsize, d+1);
			}
			
			for(int i=1; i<=r; i++) {
				for(int j=1; j<=r; j++) {
					if(i != k && j != k)
						DFW(d_a,(i-1)*newsize, (j-1)*newsize, (i-1)*newsize, (k-1)*newsize, (k-1)*newsize, (j-1)*newsize, newsize, d+1);
				}
			}	
		}
	}
}


int main(int argc, char *argv[])
{

	float *d_a;
	float *a;
	
	size_t pitch;
	rowSize = atoi(argv[1]);
	int colSize = rowSize;
	int i,j;
	hipError_t err = hipSuccess;  
	size_t totalSize = rowSize*colSize*sizeof(float);	

	a = (float *) malloc(totalSize);
	if (!a)
	{
		printf("Unable to allocate memory for host array\n");
		return 1;
	}	
	
	err = hipMallocPitch(&d_a, &pitch, rowSize * sizeof(float), colSize);
	if(!d_a)
		{
			printf("memory failed for hipMalloc");
			return 1;
		}  
  	
	if(err !=0){
        	printf("%s-%d",hipGetErrorString(err),3);
        	//getchar();  
    	}   


	for(i = 0; i < rowSize;i++)
		for (j=0;j<colSize;j++)
		{
			if (i == j){
				a[i*rowSize+j] = 0;
			}
			else {

				a[i*rowSize+j] = (i+j)%5? (i+j) : (i+j)%7;
			}
		}
 

	err = hipMemcpy(d_a, a, totalSize, hipMemcpyHostToDevice);	

	struct timeval  tv1, tv2;
	gettimeofday(&tv1, NULL);

	AFW(d_a,0,0,0,0,0,0,rowSize,0);

	gettimeofday(&tv2, NULL);
	printf ("Total Execution time = %f seconds\n", (double)(tv2.tv_usec - tv1.tv_usec) / 1000000 + (double)(tv2.tv_sec - tv1.tv_sec));

	err = hipMemcpy(a, d_a, totalSize, hipMemcpyDeviceToHost);

	print_matrix(a);

	return 0;
}

