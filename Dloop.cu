#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>

int rowSize;

__global__ void printGpu(float *d_a, int size)
{
	   int i,j;
        for(i=0;i<size;i++)
        {
                for(j=0;j<size;j++)
                        printf("%0.1f\t", d_a[i*size+j]);
               	printf("\n");
        }
}

__global__ void Dloop_FW(float *d_a,int k, int rowSize)
{

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col >= rowSize)
		return;

	__shared__ int intermed;
        if (threadIdx.x == 0) {
               	intermed = d_a[rowSize*blockIdx.y + k];
	}
        __syncthreads();


        d_a[blockIdx.y*rowSize + col]  = fmin(d_a[blockIdx.y*rowSize + col], intermed + d_a[k*rowSize+col]);

}

void print_matrix(float *d,int size)
{
	int i,j;
	for(i=0;i<size;i++)	
	{
		for(j=0;j<size;j++)
			printf("%0.1f\t", d[i*size+j]);
		puts("");
	}
}

int main(int argc, char** argv) 
{

	float *d_a;
	float *a;
	
	size_t pitch;
	rowSize = 8192;
	int colSize = rowSize;
	int i,j,k;
	hipError_t err = hipSuccess;  
	size_t totalSize = rowSize*colSize*sizeof(float);	

	a = (float *) malloc(totalSize);
	if (!a)
	{
		printf("Unable to allocate memory for host array\n");
		return 1;
	}	
	
	err = hipMallocPitch(&d_a, &pitch, rowSize * sizeof(float), colSize);
	if(!d_a)
		{
			printf("memory failed for hipMalloc");
			return 1;
		}  
  	
	if(err !=0){
        	printf("%s-%d",hipGetErrorString(err),3);
        	//getchar();  
    	}   


	for(i = 0; i < rowSize;i++)
		for (j=0;j<colSize;j++)
		{
			if (i == j){
				a[i*rowSize+j] = 0;
			}
			else {

				a[i*rowSize+j] = (i+j)%5? (i+j) : (i+j)%7;
			}
		}
 

	//puts("input matrix :");

	//print_matrix(a,rowSize);
	
	err = hipMemcpy(d_a, a, totalSize, hipMemcpyHostToDevice);

	if(err !=0){
        	printf("after h2d %s-%d",hipGetErrorString(err),3);
        getchar();  
    	}   
	

	int threadsPerBlock = 1024;
	dim3 blocksPerGrid( (colSize + threadsPerBlock - 1)/threadsPerBlock ,rowSize);

	for(k=0;k<rowSize;k++)
	{
                	Dloop_FW<<<blocksPerGrid,threadsPerBlock>>>(d_a,k,rowSize);
			hipDeviceSynchronize();
		
	}

	printf("error = %s\n", hipGetErrorString(hipGetLastError()));

       	err = hipMemcpy(a, d_a, totalSize, hipMemcpyDeviceToHost);

	if(err !=0){
        	printf("final %s-%d",hipGetErrorString(err),3);
        //	getchar();  
    	}   

	puts("output matrix :");	
	print_matrix(a,rowSize);

	free(a);
	hipFree(d_a);
	return 0;
}
