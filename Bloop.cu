#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>


int rowSize;

__global__ void printGpu(float *d_a, int size)
{
	   int i,j;
        for(i=0;i<size;i++)
        {
                for(j=0;j<size;j++)
                        printf("%0.1f\t", d_a[i*size+j]);
               	printf("\n");
        }
}

__global__ void Bloop_FW(float *d_a,int i,int k, int colSize)
{

	__shared__ int intermed;
	int col =  blockIdx.x * blockDim.x + threadIdx.x;
	if(col >= colSize)
		return;

	if (threadIdx.x == 0) {
		intermed = d_a[i*colSize+k];
		//d_a[i*colSize + threadIdx.x ]  = ((intermed + d_a[k*colSize + threadIdx.x]) < d_a[i*colSize + threadIdx.x ]) ? (intermed + d_a[k*colSize + threadIdx.x]) : (d_a[i*colSize + threadIdx.x ]);
		//d_a[i*colSize + col ]  = fmin(intermed + d_a[k*colSize + col] ,  d_a[i*colSize+col ]); 	
	}
	__syncthreads();

	d_a[i*colSize +	col ]  = fmin(intermed + d_a[k*colSize + col], d_a[i*colSize+col ]);
	//d_a[i*colSize + threadIdx.x ]  = ((intermed + d_a[k*colSize + threadIdx.x]) < d_a[i*colSize + threadIdx.x ]) ? (intermed + d_a[k*colSize + threadIdx.x]) : (d_a[i*colSize + threadIdx.x ]);


}

void print_matrix(float *d,int size)
{
	int i,j;
	for(i=0;i<size;i++)	
	{
		for(j=0;j<size;j++)
			printf("%0.1f\t", d[i*size+j]);
		puts("");
	}
}

int main(int argc, char** argv) 
{

	float *d_a;
	float *a;
	
	size_t pitch;
	rowSize = 8192;
	int colSize = rowSize;
	int i,j,k;
	hipError_t err = hipSuccess;  
	size_t totalSize = rowSize*colSize*sizeof(float);	

	a = (float *) malloc(totalSize);
	if (!a)
	{
		printf("Unable to allocate memory for host array\n");
		return 1;
	}	
	
	err = hipMallocPitch(&d_a, &pitch, rowSize * sizeof(float), colSize);
	if(!d_a)
	{
		printf("memory failed for hipMalloc");
		return 1;
	}  
  	
	if(err !=0){
        	printf("%s-%d",hipGetErrorString(err),3);
   		return 1;  
    	}   

	for(i = 0; i < rowSize;i++)
		for (j=0;j<colSize;j++)
		{
			if (i == j){
				a[i*rowSize+j] = 0;
			}
			else {

				a[i*rowSize+j] = (i+j)%5? (i+j) : (i+j)%7;
			}
		}
 

	//puts("input matrix :");

	//print_matrix(a,rowSize);
	
	err = hipMemcpy(d_a, a, totalSize, hipMemcpyHostToDevice);

	if(err !=0){
        	printf("after h2d %s-%d",hipGetErrorString(err),3);
        getchar();  
    	}   
	

	int threadsPerBlock;
	int noOfBlocks;

	if (rowSize < 1024)
        { 
                threadsPerBlock	= rowSize;
       	}
	else
	{
		threadsPerBlock = 1024;
	}

        noOfBlocks = rowSize / threadsPerBlock;


	for(k=0;k<rowSize;k++)
	{
	   for(i = 0; i < rowSize;i++)
	   {
                	Bloop_FW<<<noOfBlocks,threadsPerBlock>>>(d_a,i,k,rowSize);
			hipDeviceSynchronize();
	   }
	}

	printf("error = %s\n", hipGetErrorString(hipGetLastError()));

       	err = hipMemcpy(a, d_a, totalSize, hipMemcpyDeviceToHost);

	if(err !=0){
        	printf("final %s-%d",hipGetErrorString(err),3);
        	return 1; 
    	}   

	puts("output matrix :");	
	print_matrix(a,rowSize);

	free(a);
	hipFree(d_a);
	return 0;
}
